/*
 * cuda_device.cu
 *
 * Demo code to get the number of CUDA devices attached.
 */
#include <cstdlib>
#include <stdio.h>

int main() {
    auto devices = 0;

    hipGetDeviceCount(&devices);
    for (auto ndx = 0; ndx < devices; ndx++) {
        hipDeviceProp_t properties;
        hipGetDeviceProperties(&properties, ndx);
        printf("Device Number: %d\n", ndx);
        printf("  Device name: %s\n", properties.name);
        printf("  Memory Clock (Khz): %d\n", properties.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", properties.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.2f\n\n", 2.0 * properties.memoryClockRate * (properties.memoryBusWidth / 8) / 1.0e6);
    }

    return EXIT_SUCCESS;
}